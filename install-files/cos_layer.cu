#include "hip/hip_runtime.h"
// Cos neuron activation function layer.

#include <vector>

#include "caffe/layers/cos_layer.hpp"

namespace caffe {
  template <typename Dtype>
  __global__ void CosForward(const int n, const Dtype* in, Dtype* out) {
    CUDA_KERNEL_LOOP(index, n) {
      out[index] = cos(in[index]);
    }
  }

  template <typename Dtype>
  void CosLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    CosForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, top_data);
    CUDA_POST_KERNEL_CHECK;
  }

  template <typename Dtype>
  __global__ void CosBackward(const int n, const Dtype* in_diff,
      const Dtype* out_data, Dtype* out_diff) {
    CUDA_KERNEL_LOOP(index, n) {
      Dtype cosx = out_data[index];
      out_diff[index] = in_diff[index] * (-1.0 * sin(cosx));
    }
  }

  template <typename Dtype>
  void CosLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down,
      const vector<Blob<Dtype>*>& bottom) {
    if (propagate_down[0]) {
      const Dtype* bottom_data = bottom[0]->gpu_data();
      const Dtype* top_diff = top[0]->gpu_diff();
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      const int count = bottom[0]->count();
      // NOLINT_NEXT_LINE(whitespace/operators)
      CosBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, top_diff, bottom_data, bottom_diff);
      CUDA_POST_KERNEL_CHECK;
    }
  }

INSTANTIATE_LAYER_GPU_FUNCS(CosLayer);

}  // namespace caffe
